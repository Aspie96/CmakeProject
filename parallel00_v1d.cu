#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#ifdef __cplusplus
//#ifndef _MSC_VER
#define restrict __restrict__
//#endif
#endif

void pascal(int *p, int n) {
	n--;
	p[0] = 1;
	for(int k = 0; k < (n >> 1); k++) {
		p[k + 1] = p[k] * (n - k) / (k + 1);
	}
}

#define APPROX_DIVIDE2(A, B) (((A) >> (B)) + (((A) >> ((B) - 1)) & 1))

//#ifndef _MSC_VER
#define restrict __restrict__
//#endif

__global__
void kernel1b1(const unsigned short *restrict img, int width, int height, size_t result_pitch, size_t img_pitch, int n, unsigned short *restrict result) {
	int i, j, z, k, l, c, m;
	z = blockIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.z * blockDim.z + threadIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n >> 1; k++) {
			l = i + k - n / 2;
			m = 0;
			if(0 <= l && l < width) {
				m = img[(z * height + j) * img_pitch + l];
			}
			l = i + n - 1 - k - n / 2;
			if(0 <= l && l < width) {
				m += img[(z * height + j) * img_pitch + l];
			}
			c += m;
		}
		l = i + k - n / 2;
		if(0 <= l && l < width) {
			c += img[(z * height + j) * img_pitch + l];
		}
		result[(z * height + j) * result_pitch + i] = c / n;
	}
}

__global__
void kernel2a1(const unsigned short *restrict img, int width, int height, size_t result_pitch, size_t img_pitch, int n, const int *restrict filter, unsigned short *restrict result) {
	int i, j, z, k, l, m, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n >> 1; k++) {
			l = j + k - n / 2;
			m = 0;
			if(0 <= l && l < height) {
				m = img[(z * height + l) * img_pitch + i];
			}
			l = j + n - 1 - k - n / 2;
			if(0 <= l && l < height) {
				m += img[(z * height + l) * img_pitch + i];
			}
			c += filter[k] * m;
		}
		l = j + k - n / 2;
		if(0 <= l && l < height) {
			c += filter[k] * img[(z * height + l) * img_pitch + i];
		}
		result[(z * height + j) * result_pitch + i] = APPROX_DIVIDE2(c, n - 1);
	}
}

__global__
void kernel1b3(const unsigned short *restrict img, int width, int height, size_t result_pitch, size_t img_pitch, int n, unsigned short *restrict result) {
	int i, j, z, k, l, c, m;
	z = blockIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.z * blockDim.z + threadIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n >> 1; k++) {
			m = 0;
			l = i + k - n / 2;
			if(0 <= l && l < width) {
				m = img[(j * img_pitch + l * 3) + z];
			}
			l = i + n - 1 - k - n / 2;
			if(0 <= l && l < width) {
				m += img[(j * img_pitch + l * 3) + z];
			}
			c += m;
		}
		l = i + k - n / 2;
		if(0 <= l && l < width) {
			c += img[(j * img_pitch + l * 3) + z];
		}
		result[(j * result_pitch + i * 3) + z] = c / n;
	}
}

__global__
void kernel2a3(const unsigned short *restrict img, int width, int height, size_t result_pitch, size_t img_pitch, int n, const int *restrict filter, unsigned short *restrict result) {
	int i, j, z, k, l, c, m;
	z = blockIdx.x;
	i = blockIdx.y * blockDim.y + threadIdx.y;
	j = blockIdx.z * blockDim.z + threadIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n >> 1; k++) {
			l = j + k - n / 2;
			m = 0;
			if(0 <= l && l < width) {
				m = img[(l * img_pitch + i * 3) + z];
			}
			l = j + n - 1 - k - n / 2;
			if(0 <= l && l < width) {
				m += img[(l * img_pitch + i * 3) + z];
			}
			c += filter[k] * m;
		}
		l = j + k - n / 2;
		if(0 <= l && l < width) {
			c += filter[k] * img[(l * img_pitch + i * 3) + z];
		}
		result[(j * result_pitch + i * 3) + z] = APPROX_DIVIDE2(c, n - 1);
	}
}

void blur1(int width, int height) {
	int i;
	size_t img1_pitch, img2_pitch;
	unsigned short *restrict img1, *restrict img2;
	dim3 blocks(3, (width + 31) / 32, (height + 31) / 32);
	dim3 threadsPerBlock(1, 32, 32);

	hipMallocPitch((void **)&img1, &img1_pitch, sizeof(unsigned short) * width, height * 3);
	img1_pitch /= sizeof(unsigned short);
	hipMallocPitch((void **)&img2, &img2_pitch, sizeof(unsigned short) * width, height * 3);
	img2_pitch /= sizeof(unsigned short);
	for(i = 0; i < 1000; i++) {
		kernel1b1 << <blocks, threadsPerBlock >> > (img1, width, height, img2_pitch, img1_pitch, 17, img2);
	}
	hipFree(img1);
	hipFree(img2);
	hipDeviceSynchronize();
}

void blur3(int width, int height) {
	int i;
	size_t img1_pitch, img2_pitch;
	unsigned short *restrict img1, *restrict img2;
	dim3 blocks(3, (width + 31) / 32, (height + 31) / 32);
	dim3 threadsPerBlock(1, 32, 32);

	hipMallocPitch((void **)&img1, &img1_pitch, sizeof(unsigned short) * width * 3, height);
	img1_pitch /= sizeof(unsigned short);
	hipMallocPitch((void **)&img2, &img2_pitch, sizeof(unsigned short) * width * 3, height);
	img2_pitch /= sizeof(unsigned short);
	for(i = 0; i < 1000; i++) {
		kernel1b3 << <blocks, threadsPerBlock >> > (img1, width, height, img2_pitch, img1_pitch, 17, img2);
	}
	hipFree(img1);
	hipFree(img2);
	hipDeviceSynchronize();
}

int main(void) {
	clock_t begin, end;
	begin = clock();
	blur1(4096, 4096);
	end = clock();
	printf("Time: %f", (double)(end - begin) / CLOCKS_PER_SEC);
	return 0;
}
