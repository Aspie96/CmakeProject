#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <cstdlib>
#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb_image_resize.h"

#define S_R_SHIFT(A, B)	(((B) >= 0) ? ((A) >> (B)) : (A) << -(B))
#define APPROX_DIVIDE1(A, B) (S_R_SHIFT(A, B) + (S_R_SHIFT(A, (B) - 1) & 1))
#define APPROX_DIVIDE2(A, B) (((A) >> (B)) + (((A) >> ((B) - 1)) & 1))
#ifndef N
#define N 13
#endif
#ifndef WIDTH
#define WIDTH 0
#endif
#ifndef HEIGHT
#define HEIGHT WIDTH
#endif
#ifndef SAVED
#define SAVED (N - 1)
#endif
#define NBLOCK 8
#define NBLOCKH 2
#define NBLOCKH 2

void pascal(int *p, int n) {
	n--;
	p[0] = 1;
	for(int k = 0; k < n; k++) {
		p[k + 1] = p[k] * (n - k) / (k + 1);
	}
}

void checkCudaErrors(hipError_t error) {

}

__global__
void kernel1a(const stbi_uc *img, int width, int height, int n, int *kernel, unsigned short *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			l = i + k - n / 2;
			if(0 <= l && l < width) {
				c += kernel[k] * img[(j * width + l) * 3 + z];
			}
		}
		result[(z * height + j) * width + i] = APPROX_DIVIDE1(c, n - 9);
	}
}

__global__
void kernel1b(unsigned short *img, int width, int height, int n, int *kernel, unsigned short *result) {
	int i, j, z, k, l, c, b;
	extern __shared__ unsigned short tile[];
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	for(b = 0; b < NBLOCKH; b++) {
		i = (blockIdx.x * NBLOCKH + b) * blockDim.x + threadIdx.x;
		tile[threadIdx.y * (n - 1 + NBLOCKH * blockDim.x) + (n >> 1) + threadIdx.x + blockDim.x * b] = img[(z * height + j) * width + i];
	}
	if(!((n >> 1) <= threadIdx.x && threadIdx.x < blockDim.x - (n >> 1))) {
		/*int aux = (threadIdx.x < n >> 1) ? (blockIdx.x + 1) * NBLOCKH * blockDim.x + threadIdx.x : (blockIdx.x * NBLOCKH - 1) * blockDim.x + threadIdx.x;
		int aux2 = (threadIdx.x < n >> 1) ? (n >> 1) + blockDim.x * NBLOCKH : (n >> 1) - blockDim.x;*/
		int aux = (threadIdx.x < n >> 1) ? blockIdx.x * NBLOCKH * blockDim.x + threadIdx.x - (n >> 1) : i + (n >> 1);
		int aux2 = (threadIdx.x < n >> 1) ? 0 : n - 1 + blockDim.x * (NBLOCKH - 1);
		tile[threadIdx.y * (n - 1 + NBLOCKH * blockDim.x) + threadIdx.x + aux2] = (0 <= aux && aux < width) ? img[(z * height + j) * width + aux] : 0;
	}
	if(threadIdx.y == 0 && threadIdx.x < (n >> 1) + 1) {
		tile[blockDim.y * (blockDim.x * NBLOCKH + n - 1) + threadIdx.x] = kernel[threadIdx.x];
	}
	__syncthreads();
	for(b = 0; b < NBLOCKH; b++) {
		i = (blockIdx.x * NBLOCKH + b) * blockDim.x + threadIdx.x;
		if(i < width && j < height) {
			c = 0;
			for(k = 0; k < n >> 1; k++) {
				c += tile[blockDim.y * (blockDim.x * NBLOCKH + n - 1) + k] * (tile[threadIdx.y * (n - 1 + NBLOCKH * blockDim.x) + b * blockDim.x + threadIdx.x + k] + tile[threadIdx.y * (n - 1 + NBLOCKH * blockDim.x) + b * blockDim.x + threadIdx.x + n - 1 - k]);
			}
			c += tile[blockDim.y * (blockDim.x * NBLOCKH + n - 1) + k] * tile[threadIdx.y * (n - 1 + NBLOCKH * blockDim.x) + b * blockDim.x + threadIdx.x + k];
			result[(z * height + j) * width + i] = APPROX_DIVIDE2(c, n - 1);
		}
	}
}

__global__
void kernel2a(unsigned short *img, int width, int height, int n, int *kernel, int nblock, unsigned short *result) {
	int i, j, z, k, l, c, b;
	extern __shared__ unsigned short tile[];
	i = blockIdx.x * blockDim.x + threadIdx.x;
	z = blockIdx.z;
	for(b = 0; b < nblock; b++) {
		j = (blockIdx.y * nblock + b) * blockDim.y + threadIdx.y;
		tile[(threadIdx.y + (n >> 1) + blockDim.y * b) * blockDim.x + threadIdx.x] = img[(z * height + j) * width + i];
	}
	if(!((n >> 1) <= threadIdx.y && threadIdx.y < blockDim.y - (n >> 1))) {
		int aux = (threadIdx.y < n >> 1) ? blockIdx.y * nblock * blockDim.y + threadIdx.y - (n >> 1) : j + (n >> 1);
		int aux2 = (threadIdx.y < n >> 1) ? 0 : n - 1 + blockDim.y * (nblock - 1);
		tile[(threadIdx.y + aux2) * blockDim.x + threadIdx.x] = (0 <= aux && aux < height) ? img[(z * height + aux) * width + i] : 0;
	} else if(threadIdx.y == (n >> 1) + 1 && threadIdx.x < (n >> 1) + 1) {
		tile[blockDim.x * (blockDim.y * nblock + n - 1) + threadIdx.x] = kernel[threadIdx.x];
	}
	__syncthreads();
	for(b = 0; b < nblock; b++) {
		j = (blockIdx.y * nblock + b) * blockDim.y + threadIdx.y;
		if(i < width && j < height) {
			c = 0;
			for(k = 0; k < n >> 1; k++) {
				c += tile[blockDim.x * (blockDim.y * nblock + n - 1) + k] * (tile[(threadIdx.y + k + blockDim.y * b) * blockDim.x + threadIdx.x] + tile[(threadIdx.y + n - 1 - k + blockDim.y * b) * blockDim.x + threadIdx.x]);
			}
			c += tile[blockDim.x * (blockDim.y * nblock + n - 1) + k] * tile[(threadIdx.y + k + blockDim.y * b) * blockDim.x + threadIdx.x];
			result[(z * height + j) * width + i] = APPROX_DIVIDE2(c, n - 1);
		}
	}
}

__global__
void kernel2b(unsigned short *img, int width, int height, int n, int *kernel, stbi_uc *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			l = j + k - n / 2;
			if(0 <= l && l < height) {
				c += kernel[k] * img[(z * height + l) * width + i];
			}
		}
		result[(j * width + i) * 3 + z] = APPROX_DIVIDE2(c, n + 7);
	}
}

void applyKernel1(const stbi_uc *img, int width, int height, int n, int *kernel, int *result) {
	int i, j, k, l;
	for(i = 0; i < width; i++) {
		for(j = 0; j < height; j++) {
			result[(j * width + i) * 3] = 0;
			result[(j * width + i) * 3 + 1] = 0;
			result[(j * width + i) * 3 + 2] = 0;
			for(k = 0; k < n; k++) {
				l = i + k - n / 2;
				if(0 <= l && l < width) {
					result[(j * width + i) * 3] += kernel[k] * img[(j * width + l) * 3];
					result[(j * width + i) * 3 + 1] += kernel[k] * img[(j * width + l) * 3 + 1];
					result[(j * width + i) * 3 + 2] += kernel[k] * img[(j * width + l) * 3 + 2];
				}
			}
		}
	}
}

void applyKernel2(int *img, int width, int height, int n, int *kernel, stbi_uc *result) {
	int i, j, k, l, r, g, b;
	for(i = 0; i < width; i++) {
		for(j = 0; j < height; j++) {
			r = 0;
			g = 0;
			b = 0;
			for(k = 0; k < n; k++) {
				l = j + k - n / 2;
				if(0 <= l && l < height) {
					r += kernel[k] * img[(l * width + i) * 3];
					g += kernel[k] * img[(l * width + i) * 3 + 1];
					b += kernel[k] * img[(l * width + i) * 3 + 2];
				}
			}
			result[(j * width + i) * 3] = r >> (n - 1 + n - 1);
			result[(j * width + i) * 3 + 1] = g >> (n - 1 + n - 1);
			result[(j * width + i) * 3 + 2] = b >> (n - 1 + n - 1);
		}
	}
}

__global__
void saxpy(int n, stbi_uc *y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n) y[i] = 0;
}

void blur(int n, int width, int height, stbi_uc *img_d, unsigned short *aux1_d, unsigned short *aux2_d) {
	int *filter1;
	int *filter2;
	int *filter1_d;
	int *filter2_d;
	int n_init;
	if(n <= 15 || (n - 1) % 14 == 0) {
		n_init = 15;
	} else {
		n_init = ((n - 1) % 14) + 1;
	}
	filter1 = (int *)malloc(sizeof(int) * n_init);
	filter2 = (int *)malloc(sizeof(int) * 15);
	pascal(filter1, n_init);
	pascal(filter2, 15);
	dim3 blocks((width + 31) / 32, (height + 31) / 32, 3);
	dim3 blocks2((width + 31) / 32, (height + 31) / 32 / NBLOCK, 3);
	dim3 blocks3((width + 1024) / 1024 / NBLOCKH, height, 3);
	dim3 threadsPerBlock(32, 32, 1);
	dim3 threadsPerBlock2(1024, 1, 1);
	hipMalloc(&filter1_d, sizeof(int) * n_init);
	hipMalloc(&filter2_d, sizeof(int) * 15);
	hipMemcpy(filter1_d, filter1, sizeof(int) * n_init, hipMemcpyHostToDevice);
	hipMemcpy(filter2_d, filter2, sizeof(int) * 15, hipMemcpyHostToDevice);
	//hipError_t b = hipGetLastError();
	kernel1a << <blocks, threadsPerBlock >> > (img_d, width, height, n_init, filter1_d, aux1_d);
	hipDeviceSynchronize();
	//hipError_t dd = hipGetLastError();
	for(int i = n_init; i < (n - 1); i += 14) {
		kernel2a << <blocks2, threadsPerBlock, sizeof(unsigned short) *((32) *(32 * NBLOCK + 15 - 1) + 8) >> > (aux1_d, width, height, 15, filter2_d, NBLOCK, aux2_d);
		hipDeviceSynchronize();
		kernel1b << <blocks3, threadsPerBlock2, sizeof(unsigned short) *((32) * (32 * NBLOCK + 15 - 1) + 8) >> > (aux2_d, width, height, 15, filter2_d, aux1_d);
		hipDeviceSynchronize();
	}
	kernel2b << <blocks, threadsPerBlock >> > (aux1_d, width, height, n_init, filter1_d, img_d);
	free(filter1);
	free(filter2);
}

double test_blur_time(int n, int width, int height, stbi_uc *img_d, unsigned short *aux1_d, unsigned short *aux2_d) {
	clock_t begin = clock();
	blur(n, width, height, img_d, aux1_d, aux2_d);
	clock_t end = clock();
	return (double)(end - begin) / CLOCKS_PER_SEC;
}

int main(void) {
	printf("Parallel version - no constant memory - yes shared memory\n");
	int nk = N;
	const char fname[] = "./CmakeProject/img2.png";
	int width, height, chn;
	stbi_uc *img = stbi_load(fname, &width, &height, &chn, 3);
	stbi_uc *img_d;
	if(WIDTH != 0) {
		stbi_uc *img_r = (stbi_uc *)malloc(sizeof(stbi_uc) * WIDTH * HEIGHT * 3);
		stbir_resize_uint8(img, width, height, 0, img_r, WIDTH, HEIGHT, 0, 3);
		width = WIDTH;
		height = HEIGHT;
		img = img_r;
	}
	checkCudaErrors(hipMalloc(&img_d, sizeof(stbi_uc) * width * height * 3));
	unsigned short *aux1_d, *aux2_d;
	checkCudaErrors(hipMalloc(&aux1_d, sizeof(unsigned short) * width * height * 3));
	checkCudaErrors(hipMalloc(&aux2_d, sizeof(unsigned short) * width * height * 3));
	printf("Size of image: %dx%d\n", width, height);
	for(int i = 0; i < nk; i++) {
		checkCudaErrors(hipMemcpy(img_d, img, sizeof(stbi_uc) * width * height * 3, hipMemcpyHostToDevice));
		hipDeviceSynchronize();
		int ks = (1 << (i + 1)) + 1;
		printf("Blurring with kernel size %d...", ks);
		double time = test_blur_time(ks, width, height, img_d, aux1_d, aux2_d);
		printf(" Blurred in %f seconds!\n", time);
		if(i == SAVED) {
			checkCudaErrors(hipMemcpy(img, img_d, sizeof(stbi_uc) * width * height * 3, hipMemcpyDeviceToHost));
			//hipError_t b = hipGetLastError();
			const char fname2[] = "image2.bmp";
			stbi_write_bmp(fname2, width, height, 3, img);
		}
	}
	hipFree(aux1_d);
	hipFree(aux2_d);
	hipFree(img_d);
	hipError_t b = hipGetLastError();
	printf("\n");
	return 0;
}
