#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <cstdlib>
#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb_image_resize.h"

#define APPROX_DIVIDE(A, B) (((A) >> (B)) + (((A) >> ((B) - 1)) & 1))
#ifndef N
#define N 13
#endif
#ifndef WIDTH
#define WIDTH 0
#endif
#ifndef HEIGHT
#define HEIGHT WIDTH
#endif
#ifndef SAVED
#define SAVED (N - 1)
#endif

void pascal(int *p, int n) {
	n--;
	p[0] = 1;
	for(int k = 0; k < n; k++) {
		p[k + 1] = p[k] * (n - k) / (k + 1);
	}
}

void checkCudaErrors(hipError_t error) {

}

__global__
void kernel1a(const stbi_uc *img, int width, int height, int n, int *kernel, unsigned short *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	extern __shared__ unsigned short tile[];
	int tileW = blockDim.x + n - 1;
	int tileH = blockDim.y;
	int blockS = blockDim.x * blockDim.y;
	for(k = 0; k < (tileW * tileH) / blockS; k++) {
		int pos = k + (threadIdx.y * blockDim.x + threadIdx.x) * ((tileW * tileH) / blockS);
		int imgX = blockDim.x * blockIdx.x - n / 2 +pos % tileW;
		int imgY = blockDim.y * blockIdx.y+pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(imgY * width + imgX) * 3 + z] << 8 : 0;
	}
	int pos = blockDim.x * blockDim.y * k + threadIdx.y * blockDim.x + threadIdx.x;
	if(pos < tileW * tileH) {
		int imgX = blockDim.x * blockIdx.x - n / 2 + pos % tileW;
		int imgY = blockDim.y * blockIdx.y+ pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(imgY * width + imgX) * 3 + z] << 8 : 0;
	}
	__syncthreads();
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			c += kernel[k] * tile[(threadIdx.y) * tileW + threadIdx.x + k];
		}
		result[(z * height + j) * width + i] = APPROX_DIVIDE(c, n - 1);
	}
}

__global__
void kernel1b(unsigned short *img, int width, int height, int n, int *kernel, unsigned short *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	extern __shared__ unsigned short tile[];
	int tileW = blockDim.x + n - 1;
	int tileH = blockDim.y;
	int blockS = blockDim.x * blockDim.y;
	for(k = 0; k < (tileW * tileH) / blockS; k++) {
		int pos = k + (threadIdx.y * blockDim.x + threadIdx.x) * ((tileW * tileH) / blockS);
		int imgX = blockDim.x * blockIdx.x - n / 2 + pos % tileW;
		int imgY = blockDim.y * blockIdx.y + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	int pos = blockDim.x * blockDim.y * k + threadIdx.y * blockDim.x + threadIdx.x;
	if(pos < tileW * tileH) {
		int imgX = blockDim.x * blockIdx.x - n / 2 + pos % tileW;
		int imgY = blockDim.y * blockIdx.y + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	__syncthreads();
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			c += kernel[k] * tile[(threadIdx.y) * tileW + threadIdx.x + k];
		}
		result[(z * height + j) * width + i] = APPROX_DIVIDE(c, n - 1);
	}
}

__global__
void kernel2a(unsigned short *img, int width, int height, int n, int *kernel, unsigned short *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	extern __shared__ unsigned short tile[];
	int tileW = blockDim.x;
	int tileH = blockDim.y + n - 1;
	int blockS = blockDim.x * blockDim.y;
	for(k = 0; k < (tileW * tileH) / blockS; k++) {
		int pos = k + (threadIdx.y * blockDim.x + threadIdx.x) * ((tileW * tileH) / blockS);
		int imgX = blockDim.x * blockIdx.x + pos % tileW;
		int imgY = blockDim.y * blockIdx.y - n / 2 + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	int pos = blockDim.x * blockDim.y * k + threadIdx.y * blockDim.x + threadIdx.x;
	if(pos < tileW * tileH) {
		int imgX = blockDim.x * blockIdx.x + pos % tileW;
		int imgY = blockDim.y * blockIdx.y - n / 2 + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	__syncthreads();
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			c += kernel[k] * tile[(threadIdx.y + k) * tileW + (threadIdx.x)];
		}
		result[(z * height + j) * width + i] = APPROX_DIVIDE(c, n - 1);
	}
}

__global__
void kernel2b(unsigned short *img, int width, int height, int n, int *kernel, stbi_uc *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	extern __shared__ unsigned short tile[];
	int tileW = blockDim.x;
	int tileH = blockDim.y + n - 1;
	int blockS = blockDim.x * blockDim.y;
	for(k = 0; k < (tileW * tileH) / blockS; k++) {
		int pos = k + (threadIdx.y * blockDim.x + threadIdx.x) * ((tileW * tileH) / blockS);
		int imgX = blockDim.x * blockIdx.x+ pos % tileW;
		int imgY = blockDim.y * blockIdx.y - n / 2 + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	int pos = blockDim.x * blockDim.y * k + threadIdx.y * blockDim.x + threadIdx.x;
	if(pos < tileW * tileH) {
		int imgX = blockDim.x * blockIdx.x + pos % tileW;
		int imgY = blockDim.y * blockIdx.y - n / 2 + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	__syncthreads();
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			c += kernel[k] * tile[(threadIdx.y + k) * tileW + (threadIdx.x)];
		}
		result[(j * width + i) * 3 + z] = APPROX_DIVIDE(c, n + 7);
	}
}

void applyKernel1(const stbi_uc *img, int width, int height, int n, int *kernel, int *result) {
	int i, j, k, l;
	for(i = 0; i < width; i++) {
		for(j = 0; j < height; j++) {
			result[(j * width + i) * 3] = 0;
			result[(j * width + i) * 3 + 1] = 0;
			result[(j * width + i) * 3 + 2] = 0;
			for(k = 0; k < n; k++) {
				l = i + k - n / 2;
				if(0 <= l && l < width) {
					result[(j * width + i) * 3] += kernel[k] * img[(j * width + l) * 3];
					result[(j * width + i) * 3 + 1] += kernel[k] * img[(j * width + l) * 3 + 1];
					result[(j * width + i) * 3 + 2] += kernel[k] * img[(j * width + l) * 3 + 2];
				}
			}
		}
	}
}

void applyKernel2(int *img, int width, int height, int n, int *kernel, stbi_uc *result) {
	int i, j, k, l, r, g, b;
	for(i = 0; i < width; i++) {
		for(j = 0; j < height; j++) {
			r = 0;
			g = 0;
			b = 0;
			for(k = 0; k < n; k++) {
				l = j + k - n / 2;
				if(0 <= l && l < height) {
					r += kernel[k] * img[(l * width + i) * 3];
					g += kernel[k] * img[(l * width + i) * 3 + 1];
					b += kernel[k] * img[(l * width + i) * 3 + 2];
				}
			}
			result[(j * width + i) * 3] = r >> (n - 1 + n - 1);
			result[(j * width + i) * 3 + 1] = g >> (n - 1 + n - 1);
			result[(j * width + i) * 3 + 2] = b >> (n - 1 + n - 1);
		}
	}
}

__global__
void saxpy(int n, stbi_uc *y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n) y[i] = 0;
}

void blur(int n, int width, int height, stbi_uc *img_d, unsigned short *aux1_d, unsigned short *aux2_d) {
	int *filter1;
	int *filter2;
	int *filter1_d;
	int *filter2_d;
	int n_init;
	if(n <= 15 || (n - 1) % 14 == 0) {
		n_init = 15;
	} else {
		n_init = ((n - 1) % 14) + 1;
	}
	filter1 = (int *)malloc(sizeof(int) * n_init);
	filter2 = (int *)malloc(sizeof(int) * 15);
	pascal(filter1, n_init);
	pascal(filter2, 15);
	dim3 blocks((width + 31) / 32, (height + 31) / 32, 3);
	dim3 threadsPerBlock(32, 32, 1);
	hipMalloc(&filter1_d, sizeof(int) * n_init);
	hipMalloc(&filter2_d, sizeof(int) * 15);
	hipMemcpy(filter1_d, filter1, sizeof(int) * n_init, hipMemcpyHostToDevice);
	hipMemcpy(filter2_d, filter2, sizeof(int) * 15, hipMemcpyHostToDevice);
	//hipError_t b = hipGetLastError();
	kernel1a << <blocks, threadsPerBlock, sizeof(int) *(32 + n_init / 2) *(32 + n_init / 2) >> > (img_d, width, height, n_init, filter1_d, aux1_d);
	hipDeviceSynchronize();
	//hipError_t dd = hipGetLastError();
	for(int i = n_init; i < (n - 1); i += 14) {
		kernel2a << <blocks, threadsPerBlock, sizeof(int) *(32 + 15 / 2) *(32 + 15 / 2) >> > (aux1_d, width, height, 15, filter2_d, aux2_d);
		hipDeviceSynchronize();
		kernel1b << <blocks, threadsPerBlock, sizeof(int) *(32 + 15 / 2) *(32 + 15 / 2) >> > (aux2_d, width, height, 15, filter2_d, aux1_d);
		hipDeviceSynchronize();
	}
	kernel2b<<<blocks, threadsPerBlock, sizeof(int) *(32 + n_init / 2) *(32 + n_init / 2) >>>(aux1_d, width, height, n_init, filter1_d, img_d);
	free(filter1);
	free(filter2);
}

double test_blur_time(int n, int width, int height, stbi_uc *img_d, unsigned short *aux1_d, unsigned short *aux2_d) {
	clock_t begin = clock();
	blur(n, width, height, img_d, aux1_d, aux2_d);
	clock_t end = clock();
	return (double)(end - begin) / CLOCKS_PER_SEC;
}

int main(void) {
	printf("Parallel version - no constant memory - yes shared memory\n");
	int nk = N;
	const char fname[] = "../../../img2.png";
	int width, height, chn;
	stbi_uc *img = stbi_load(fname, &width, &height, &chn, 3);
	stbi_uc *img_d;
	if(WIDTH != 0) {
		stbi_uc *img_r = (stbi_uc*)malloc(sizeof(stbi_uc) * WIDTH * HEIGHT * 3);
		stbir_resize_uint8(img, width, height, 0, img_r, WIDTH, HEIGHT, 0, 3);
		width = WIDTH;
		height = HEIGHT;
		img = img_r;
	}
	checkCudaErrors(hipMalloc(&img_d, sizeof(stbi_uc) * width * height * 3));
	unsigned short *aux1_d, *aux2_d;
	checkCudaErrors(hipMalloc(&aux1_d, sizeof(unsigned short) * width * height * 3));
	checkCudaErrors(hipMalloc(&aux2_d, sizeof(unsigned short) * width * height * 3));
	printf("Size of image: %dx%d\n", width, height);
	for(int i = 0; i < nk; i++) {
		checkCudaErrors(hipMemcpy(img_d, img, sizeof(stbi_uc) * width * height * 3, hipMemcpyHostToDevice));
		hipDeviceSynchronize();
		int ks = (1 << (i + 1)) + 1;
		printf("Blurring with kernel size %d...", ks);
		double time = test_blur_time(ks, width, height, img_d, aux1_d, aux2_d);
		printf(" Blurred in %f seconds!\n", time);
		if(i == SAVED) {
			checkCudaErrors(hipMemcpy(img, img_d, sizeof(stbi_uc) * width * height * 3, hipMemcpyDeviceToHost));
			//hipError_t b = hipGetLastError();
			const char fname2[] = "image2.bmp";
			stbi_write_bmp(fname2, width, height, 3, img);
		}
	}
	hipFree(aux1_d);
	hipFree(aux2_d);
	hipFree(img_d);
	hipError_t b = hipGetLastError();
	printf("\n");
	return 0;
}
