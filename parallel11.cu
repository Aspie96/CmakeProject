#include "hip/hip_runtime.h"
﻿#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <cstdlib>
#define STB_IMAGE_RESIZE_IMPLEMENTATION
#include "stb_image_resize.h"

#define S_R_SHIFT(A, B)	(((B) >= 0) ? ((A) >> (B)) : (A) << -(B))
#define APPROX_DIVIDE1(A, B) (S_R_SHIFT(A, B) + (S_R_SHIFT(A, (B) - 1) & 1))
#define APPROX_DIVIDE2(A, B) (((A) >> (B)) + (((A) >> ((B) - 1)) & 1))
#ifndef N
#define N 13
#endif
#ifndef WIDTH
#define WIDTH 0
#endif
#ifndef HEIGHT
#define HEIGHT WIDTH
#endif
#ifndef SAVED
#define SAVED (N - 1)
#endif

__constant__ int filter1_d[15];
__constant__ int filter2_d[15];

void pascal(int *p, int n) {
	n--;
	p[0] = 1;
	for(int k = 0; k < n; k++) {
		p[k + 1] = p[k] * (n - k) / (k + 1);
	}
}

void checkCudaErrors(hipError_t error) {

}

__global__
void kernel1a(const stbi_uc *img, int width, int height, int n, unsigned short *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			l = i + k - n / 2;
			if(0 <= l && l < width) {
				c += filter1_d[k] * img[(j * width + l) * 3 + z];
			}
		}
		result[(z * height + j) * width + i] = APPROX_DIVIDE1(c, n - 9);
	}
}

__global__
void kernel1b(unsigned short *img, int width, int height, int n, unsigned short *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			l = i + k - n / 2;
			if(0 <= l && l < width) {
				c += filter2_d[k] * img[(z * height + j) * width + l];
			}
		}
		result[(z * height + j) * width + i] = APPROX_DIVIDE2(c, n - 1);
	}
}

__global__
void kernel2a(unsigned short *img, int width, int height, int n, unsigned short *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	extern __shared__ unsigned short tile[];
	int tileW = blockDim.x;
	int tileH = blockDim.y + n - 1;
	int blockS = blockDim.x * blockDim.y;
	int d = threadIdx.y * blockDim.x + threadIdx.x;
	int a = (tileW * tileH) / blockS;
	int b = d * a;
	int pos;
	for(k = 0; k < a; k++) {
		pos = k + b;
		int imgX = blockDim.x * blockIdx.x + pos % tileW;
		int imgY = blockDim.y * blockIdx.y - n / 2 + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	pos = blockS * k + d;
	if(pos < tileW * tileH) {
		int imgX = blockDim.x * blockIdx.x + pos % tileW;
		int imgY = blockDim.y * blockIdx.y - n / 2 + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	__syncthreads();
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			c += filter2_d[k] * tile[(threadIdx.y + k) * tileW + (threadIdx.x)];
		}
		result[(z * height + j) * width + i] = APPROX_DIVIDE2(c, n - 1);
	}
}

__global__
void kernel2b(unsigned short *img, int width, int height, int n, stbi_uc *result) {
	int i, j, z, k, l, c;
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = blockIdx.y * blockDim.y + threadIdx.y;
	z = blockIdx.z;
	extern __shared__ unsigned short tile[];
	int tileW = blockDim.x;
	int tileH = blockDim.y + n - 1;
	int blockS = blockDim.x * blockDim.y;
	int d = threadIdx.y * blockDim.x + threadIdx.x;
	int a = (tileW * tileH) / blockS;
	int b = d * a;
	int pos;
	for(k = 0; k < a; k++) {
		pos = k + b;
		int imgX = blockDim.x * blockIdx.x + pos % tileW;
		int imgY = blockDim.y * blockIdx.y - n / 2 + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	pos = blockS * k + d;
	if(pos < tileW * tileH) {
		int imgX = blockDim.x * blockIdx.x + pos % tileW;
		int imgY = blockDim.y * blockIdx.y - n / 2 + pos / tileW;
		tile[pos] = (0 <= imgX && width > imgX && 0 <= imgY && height > imgY) ? img[(z * height + imgY) * width + imgX] : 0;
	}
	__syncthreads();
	if(i < width && j < height) {
		c = 0;
		for(k = 0; k < n; k++) {
			c += filter1_d[k] * tile[(threadIdx.y + k) * tileW + (threadIdx.x)];
		}
		result[(z * height + j) * width + i] = APPROX_DIVIDE2(c, n - 1);
	}
}

void applyKernel1(const stbi_uc *img, int width, int height, int n, int *kernel, int *result) {
	int i, j, k, l;
	for(i = 0; i < width; i++) {
		for(j = 0; j < height; j++) {
			result[(j * width + i) * 3] = 0;
			result[(j * width + i) * 3 + 1] = 0;
			result[(j * width + i) * 3 + 2] = 0;
			for(k = 0; k < n; k++) {
				l = i + k - n / 2;
				if(0 <= l && l < width) {
					result[(j * width + i) * 3] += kernel[k] * img[(j * width + l) * 3];
					result[(j * width + i) * 3 + 1] += kernel[k] * img[(j * width + l) * 3 + 1];
					result[(j * width + i) * 3 + 2] += kernel[k] * img[(j * width + l) * 3 + 2];
				}
			}
		}
	}
}

void applyKernel2(int *img, int width, int height, int n, int *kernel, stbi_uc *result) {
	int i, j, k, l, r, g, b;
	for(i = 0; i < width; i++) {
		for(j = 0; j < height; j++) {
			r = 0;
			g = 0;
			b = 0;
			for(k = 0; k < n; k++) {
				l = j + k - n / 2;
				if(0 <= l && l < height) {
					r += kernel[k] * img[(l * width + i) * 3];
					g += kernel[k] * img[(l * width + i) * 3 + 1];
					b += kernel[k] * img[(l * width + i) * 3 + 2];
				}
			}
			result[(j * width + i) * 3] = r >> (n - 1 + n - 1);
			result[(j * width + i) * 3 + 1] = g >> (n - 1 + n - 1);
			result[(j * width + i) * 3 + 2] = b >> (n - 1 + n - 1);
		}
	}
}

__global__
void saxpy(int n, stbi_uc *y) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < n) y[i] = 0;
}

struct SimpleStruct {
	int a;
	float b;
};

__constant__ int variable[2];

__global__ void kernel() {
	printf("%d %f\n", filter1_d[0], filter1_d[1]);
}

void blur(int n, int width, int height, stbi_uc *img_d, unsigned short *aux1_d, unsigned short *aux2_d) {
	int *filter1;
	int *filter2;
	int n_init;
	if(n <= 15 || (n - 1) % 14 == 0) {
		n_init = 15;
	} else {
		n_init = ((n - 1) % 14) + 1;
	}
	filter1 = (int*)malloc(sizeof(int) * n_init);
	filter2 = (int*)malloc(sizeof(int) * 15);
	pascal(filter1, n_init);
	pascal(filter2, 15);
	dim3 blocks((width + 31) / 32, (height + 31) / 32, 3);
	dim3 threadsPerBlock(32, 32, 1);
	hipMemcpyToSymbol(HIP_SYMBOL(filter1_d), filter1, sizeof(int) * n_init);
	hipMemcpyToSymbol(HIP_SYMBOL(filter2_d), filter2, sizeof(int) * 15);
	//hipError_t b = hipGetLastError();
	kernel1a<<<blocks, threadsPerBlock>>>(img_d, width, height, n_init, aux1_d);
	hipDeviceSynchronize();
	//hipError_t dd = hipGetLastError();
	for(int i = n_init; i < (n - 1); i += 14) {
		kernel2a<<<blocks, threadsPerBlock, sizeof(short) *(32) *(32 + 15) >>>(aux1_d, width, height, 15, aux2_d);
		hipDeviceSynchronize();
		kernel1b<<<blocks, threadsPerBlock >>>(aux2_d, width, height, 15, aux1_d);
		hipDeviceSynchronize();
	}
	kernel2b<<<blocks, threadsPerBlock, sizeof(short) *(32) *(32 + 15) >>>(aux1_d, width, height, n_init, img_d);
	free(filter1);
	free(filter2);
}

double test_blur_time(int n, int width, int height, stbi_uc *img_d, unsigned short *aux1_d, unsigned short *aux2_d) {
	clock_t begin = clock();
	blur(n, width, height, img_d, aux1_d, aux2_d);
	clock_t end = clock();
	return (double)(end - begin) / CLOCKS_PER_SEC;
}

int main(void) {
	printf("Parallel version - yes constant memory - yes shared memory\n");
	int nk = N;
	const char fname[] = "./CmakeProject/img2.png";
	int width, height, chn;
	stbi_uc *img = stbi_load(fname, &width, &height, &chn, 3);
	stbi_uc *img_d;
	if(WIDTH != 0) {
		stbi_uc *img_r = (stbi_uc*)malloc(sizeof(stbi_uc) * WIDTH * HEIGHT * 3);
		stbir_resize_uint8(img, width, height, 0, img_r, WIDTH, HEIGHT, 0, 3);
		width = WIDTH;
		height = HEIGHT;
		img = img_r;
	}
	checkCudaErrors(hipMalloc(&img_d, sizeof(stbi_uc) * width * height * 3));
	unsigned short *aux1_d, *aux2_d;
	checkCudaErrors(hipMalloc(&aux1_d, sizeof(unsigned short) * width * height * 3));
	checkCudaErrors(hipMalloc(&aux2_d, sizeof(unsigned short) * width * height * 3));
	printf("Size of image: %dx%d\n", width, height);
	for(int i = 0; i < nk; i++) {
		checkCudaErrors(hipMemcpy(img_d, img, sizeof(stbi_uc) * width * height * 3, hipMemcpyHostToDevice));
		hipDeviceSynchronize();
		int ks = (1 << (i + 1)) + 1;
		printf("Blurring with kernel size %d...", ks);
		double time = test_blur_time(ks, width, height, img_d, aux1_d, aux2_d);
		printf(" Blurred in %f seconds!\n", time);
		if(i == SAVED) {
			checkCudaErrors(hipMemcpy(img, img_d, sizeof(stbi_uc) * width * height * 3, hipMemcpyDeviceToHost));
			//hipError_t b = hipGetLastError();
			const char fname2[] = "image2.bmp";
			stbi_write_bmp(fname2, width, height, 3, img);
		}
	}
	hipFree(aux1_d);
	hipFree(aux2_d);
	hipFree(img_d);
	hipError_t b = hipGetLastError();
	printf("\n");
	return 0;
}
